#include "hip/hip_runtime.h"
#include <iostream>
#include <iterator>
#include <tuple>
#include <vector>

#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>

#include "../util/cudautil.hpp"
#include "../util/util.hpp"

__device__
inline auto divergence_count(const thrust::complex<float>& c) noexcept {
    auto z = thrust::complex(0.0f, 0.0f);

    for (auto i = 0; i < 100; ++i) {
        if (std::isinf(z.real())) {
            return i;
        }

        z = z * z + c;
    }

    return 0;
}

inline std::tuple<thrust::host_vector<int>, std::size_t, std::size_t> mandelbrot_set(float real_min, float real_max, std::size_t real_size, float imag_min, float imag_max, std::size_t imag_size) noexcept {
    auto result_device = thrust::device_vector<int>(real_size * imag_size);
    auto it = thrust::counting_iterator<std::size_t>(0);
    thrust::transform(it, it + real_size * imag_size, std::begin(result_device),
        [=] __device__ (const auto& i) {
            const auto imag = imag_min + (imag_max - imag_min) / static_cast<float>(imag_size - 1) * static_cast<float>(i / imag_size);
            const auto real = real_min + (real_max - real_min) / static_cast<float>(real_size - 1) * static_cast<float>(i % imag_size);

            return divergence_count(thrust::complex(real, imag));
        }
    );

    return std::make_tuple(thrust::host_vector<int>(result_device), real_size, imag_size);
}

inline auto mandelbrot_set(float real_min, float real_max, float imag_min, float imag_max) {
    const auto [real_size, imag_size] = [&]() {
        const auto real_diff = real_max - real_min;
        const auto imag_diff = imag_max - imag_min;

        return std::make_tuple(
            static_cast<std::size_t>(1000 * std::min(real_diff / imag_diff, 1.0f)),
            static_cast<std::size_t>(1000 * std::min(imag_diff / real_diff, 1.0f))
        );
    }();

    return mandelbrot_set(real_min, real_max, real_size, imag_min, imag_max, imag_size);
}

int main(int argc, char** argv) {
    first_cudaMalloc_is_too_slow();
    cuda_check(hipDeviceSynchronize());

    util::timeit([&]() {
        mandelbrot_set(-2.0f, 2.0f, -2.0f, 2.0f);
    });

    const auto [v, w, h] = mandelbrot_set(-2.0f, 2.0f, -2.0f, 2.0f);

    {
        auto it = std::begin(v);

        for (auto i = 0_z; i < h; ++i, it += w) {
            std::copy(it, it + w, std::ostream_iterator<float>(std::cout, "\t"));
            std::cout << std::endl;
        }
    }

    return 0;
}
