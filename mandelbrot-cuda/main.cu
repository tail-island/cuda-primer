#include "hip/hip_runtime.h"
#include <iostream>
#include <iterator>
#include <tuple>
#include <vector>

#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include "../util/cudautil.hpp"
#include "../util/util.hpp"

__device__
inline auto divergence_count(const thrust::complex<float>& c) noexcept {
    auto z = thrust::complex(0.0f, 0.0f);

    for (auto i = 0; i < 100; ++i) {
        if (std::isinf(z.real())) {
            return i;
        }

        z = z * z + c;
    }

    return 0;
}

__global__
void mandelbrot_set(float real_min, float real_max, std::size_t real_size, float imag_min, float imag_max, std::size_t imag_size, int* result) {
    const auto i = blockDim.y * blockIdx.y + threadIdx.y;
    const auto j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= imag_size || j >= real_size) {  // imag_indexがimag_sizeを超えることはないのですけど、パターン化しておいたほうがバグが減るので。
        return;
    }

    const auto imag = imag_min + (imag_max - imag_min) / static_cast<float>(imag_size - 1) * static_cast<float>(i);
    const auto real = real_min + (real_max - real_min) / static_cast<float>(real_size - 1) * static_cast<float>(j);

    result[i * real_size + j] = divergence_count(thrust::complex(real, imag));
}

inline auto mandelbrot_set(float real_min, float real_max, std::size_t real_size, float imag_min, float imag_max, std::size_t imag_size) noexcept {
    auto result = std::vector<int>(real_size * imag_size);

    int* result_device;
    cuda_check(hipMalloc(&result_device, sizeof(int) * real_size * imag_size));

    const auto grid  = dim3((real_size + 256 - 1) / 256, imag_size);
    const auto block = dim3(256, 1);
    mandelbrot_set<<<grid, block>>>(real_min, real_max, real_size, imag_min, imag_max, imag_size, result_device);
    cuda_check(hipGetLastError());

    cuda_check(hipMemcpy(result.data(), result_device, sizeof(int) * real_size * imag_size, hipMemcpyDeviceToHost));

    cuda_check(hipFree(result_device));

    return std::make_tuple(result, real_size, imag_size);
}

inline auto mandelbrot_set(float real_min, float real_max, float imag_min, float imag_max) noexcept {
    const auto [real_size, imag_size] = [&]() {
        const auto real_diff = real_max - real_min;
        const auto imag_diff = imag_max - imag_min;

        return std::make_tuple(
            static_cast<std::size_t>(1000 * std::min(real_diff / imag_diff, 1.0f)),
            static_cast<std::size_t>(1000 * std::min(imag_diff / real_diff, 1.0f))
        );
    }();

    return mandelbrot_set(real_min, real_max, real_size, imag_min, imag_max, imag_size);
}

int main(int argc, char** argv) {
    first_cudaMalloc_is_too_slow();
    cuda_check(hipDeviceSynchronize());

    util::timeit([&]() {
        mandelbrot_set(-2.0f, 2.0f, -2.0f, 2.0f);
    });

    const auto [v, w, h] = mandelbrot_set(-2.0f, 2.0f, -2.0f, 2.0f);

    {
        auto it = std::begin(v);

        for (auto i = 0_z; i < h; ++i, it += w) {
            std::copy(it, it + w, std::ostream_iterator<float>(std::cout, "\t"));
            std::cout << std::endl;
        }
    }

    return 0;
}
