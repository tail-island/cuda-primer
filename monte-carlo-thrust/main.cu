#include "hip/hip_runtime.h"
#include <iostream>
#include <random>

#include <hiprand.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#include "../util/cudautil.hpp"
#include "../util/util.hpp"

struct is_in_circle {
    __device__
    bool operator()(const thrust::tuple<float, float>& p) const noexcept {
        return std::pow(thrust::get<0>(p), 2) + std::pow(thrust::get<1>(p), 2) <= 1.0f;
    }
};

inline auto monte_carlo_pi(int n, unsigned long seed) noexcept {
    auto rng = hiprandGenerator_t();
    // curand_check(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_MT19937));
    curand_check(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_MTGP32));
    curand_check(hiprandSetPseudoRandomGeneratorSeed(rng, seed));

    auto xs = thrust::device_vector<float>(n);
    curand_check(hiprandGenerateUniform(rng, xs.data().get(), n));

    auto ys = thrust::device_vector<float>(n);
    curand_check(hiprandGenerateUniform(rng, ys.data().get(), n));

    auto it = thrust::make_zip_iterator(thrust::make_tuple(std::begin(xs), std::begin(ys)));
    auto c  = thrust::count_if(it, it + n, is_in_circle());

    return 4.0f * static_cast<float>(c) / static_cast<float>(n);
}

int main(int argc, char** argv) {
    first_cudaMalloc_is_too_slow();

    cuda_check(hipDeviceSynchronize());

    util::timeit([]() {
        monte_carlo_pi(100'000'000, 0ul);
    });

    std::cout << monte_carlo_pi(100'000'000, 0ul) << std::endl;

    return 0;
}
