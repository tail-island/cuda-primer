#include "hip/hip_runtime.h"
#include <complex>
#include <iostream>
#include <iterator>
#include <tuple>
#include <vector>

#include <thrust/complex.h>

#include "../util/cudautil.hpp"
#include "../util/util.hpp"

__device__
inline auto divergence_count(const thrust::complex<float>& c) noexcept {
    auto z = thrust::complex(0.0f, 0.0f);

    for (auto i = 0; i < 100; ++i) {
        if (std::isinf(z.real())) {
            return i;
        }

        z = z * z + c;
    }

    return 0;
}

__device__
inline auto linspace(float start, float stop, std::size_t size) noexcept {
    auto result = new float[size];

    auto delta = (stop - start) / static_cast<float>(size - 1);

    for (auto i = static_cast<std::size_t>(0); i < size; ++i) {
        result[i] = start + delta * i;
    }

    return result;
}

__global__
void mandelbrot_set(float real_min, float real_max, std::size_t real_size, float imag_min, float imag_max, std::size_t imag_size, int* result) {
    auto imags = linspace(imag_min, imag_max, imag_size);
    auto reals = linspace(real_min, real_max, real_size);

    for (auto i = 0; i < imag_size; ++i) {
        for (auto j = 0; j < real_size; ++j) {
            result[i * real_size + j] = divergence_count(thrust::complex(reals[j], imags[i]));
        }
    }

    delete reals;
    delete imags;
}

inline auto mandelbrot_set(float real_min, float real_max, float imag_min, float imag_max) noexcept {
    auto [real_size, imag_size] = [&]() {
        auto real_diff = real_max - real_min;
        auto imag_diff = imag_max - imag_min;

        return std::make_tuple(
            static_cast<std::size_t>(1024 * std::min(real_diff / imag_diff, 1.0f)),
            static_cast<std::size_t>(1024 * std::min(imag_diff / real_diff, 1.0f))
        );
    }();

    auto result = std::vector<int>(real_size * imag_size);

    int* result_device;
    cuda_check(hipMalloc(&result_device, sizeof(int) * real_size * imag_size));

    mandelbrot_set<<<1, 1>>>(real_min, real_max, real_size, imag_min, imag_max, imag_size, result_device);
    cuda_check(hipGetLastError());

    cuda_check(hipMemcpy(result.data(), result_device, sizeof(int) * real_size * imag_size, hipMemcpyDeviceToHost));

    cuda_check(hipFree(result_device));

    return std::make_tuple(result, real_size, imag_size);
}

int main(int argc, char** argv) {
    first_cudaMalloc_is_too_slow();
    cuda_check(hipDeviceSynchronize());

    util::timeit([&]() {
        mandelbrot_set(-2.0f, 2.0f, -2.0f, 2.0f);
    });

    auto [v, w, h] = mandelbrot_set(-2.0f, 2.0f, -2.0f, 2.0f);

    {
        auto it = std::begin(v);

        for (auto i = 0_z; i < h; ++i, it += w) {
            std::copy(it, it + w, std::ostream_iterator<float>(std::cout, "\t"));
            std::cout << std::endl;
        }
    }

    return 0;
}
