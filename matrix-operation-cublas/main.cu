#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "../util/cudautil.hpp"
#include "../util/util.hpp"

const std::size_t M = 1001;
const std::size_t N = 1002;
const std::size_t K = 1003;

template <typename T>
inline auto transpose(std::vector<T> matrix, std::size_t row_size, std::size_t col_size) noexcept {
    auto result = std::vector<T>(col_size * row_size);

    for (auto i = 0_z; i < col_size; ++i) {
        for (auto j = 0_z; j < row_size; ++j) {
            result[i * row_size + j] = matrix[j * col_size + i];
        }
    }

    return result;
}

int main(int argc, char** argv) {
    hipblasHandle_t handle;
    cublas_check(hipblasCreate(&handle));

    auto a = transpose(util::linspace(0.0f, 1.0f, M * K), M, K);

    float* a_device;
    cuda_check(hipMalloc(&a_device, sizeof(float) * M * K));
    cublas_check(hipblasSetMatrix(M, K, sizeof(float), a.data(), M, a_device, M));

    auto b = transpose(util::linspace(0.0f, 1.0f, K * N), K, N);

    float* b_device;
    cuda_check(hipMalloc(&b_device, sizeof(float) * K * N));
    cublas_check(hipblasSetMatrix(K, N, sizeof(float), b.data(), K, b_device, K));

    float* c_device;
    cuda_check(hipMalloc(&c_device, sizeof(float) * M * N));

    cuda_check(hipDeviceSynchronize());

    auto alpha = 1.0f;
    auto beta  = 0.0f;

    util::timeit([&]() {
        cublas_check(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a_device, M, b_device, K, &beta, c_device, M));

        cuda_check(hipDeviceSynchronize());
    });

    cublas_check(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a_device, M, b_device, K, &beta, c_device, M));

    auto c = std::vector<float>(N * M, 0.0f);
    cublas_check(hipblasGetMatrix(M, N, sizeof(float), c_device, M, c.data(), M));

    cuda_check(hipFree(c_device));
    cuda_check(hipFree(b_device));
    cuda_check(hipFree(a_device));

    cublas_check(hipblasDestroy(handle));

    std::cout << c[0 * M] << std::endl;
    std::cout << c[1 * M] << std::endl;
    std::cout << c[2 * M] << std::endl;

    std::cout << c[N * M - 1 - 2 * M] << std::endl;
    std::cout << c[N * M - 1 - 1 * M] << std::endl;
    std::cout << c[N * M - 1 - 0 * M] << std::endl;

    return 0;
}
